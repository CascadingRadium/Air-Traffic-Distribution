#include "hip/hip_runtime.h"
#include<iostream>
#include<vector>
#include<fstream>
#include "hip/hip_runtime_api.h"
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#define watch(x) cout << boolalpha << (#x) << " is " << (x) <<'\n'
#define watcharr(x) for(auto i:x)cout<<i<<' ';cout<<'\n';
#define NumThreads 32
#define MaxPathLen 1250
#define PI 3.141592653589793238
const double RadConvFactorToMultiply=180/PI;
using namespace std;
#include "GeneticAlgorithm.h"

int main()
{	
	/*Input File Name*/
	string InputFileName="InputFromFrontend.txt";

	/*Output File Name*/
	string OutputFileName="OutputToSimulator.txt";
	
	/*Supplementary Files */
	string GraphFileName="CppGraph.txt";
	string CentroidFileName="CppCentroids.txt";
	
	/* GA Parameters*/
	int NumSectors=1250;
	int PopulationSize=4000;
	int SelectionSize=2000;
	int NumberOfMutations=1;
	int NumberOfGenerations=50;
	
	
	/* Read OD Pairs */
	vector<pair<int,int>> ODPairs;
	readInput(ODPairs,InputFileName);
	
	/* Call CUDA Genetic Algorithm to solve the Congestion Game*/
	int NumODPairs=ODPairs.size();
	int Paths[NumODPairs][MaxPathLen];
	getPaths(ODPairs,Paths,NumSectors,PopulationSize,SelectionSize,NumberOfMutations,NumberOfGenerations,GraphFileName,CentroidFileName);// Input,Output
	
	/*Output all Paths to Output File*/
	writeOutput(Paths,OutputFileName,NumODPairs);
	cout<<'\n';
	return 0;
}

void writeOutput(int Paths[][MaxPathLen], string OutputFileName, int NumODPairs)
{
	ofstream file(OutputFileName);
	string line="";
	for(int i=0;i<NumODPairs;i++)
	{
		line="";
		for(int j=0;j<MaxPathLen;j++)
		{
			if(Paths[i][j]!=-1)
				line+=to_string(Paths[i][j])+",";
			else
				break;
		}
		if(line.length()>0)
			line.pop_back();
		line.push_back('\n');
		file<<line;
	}
	file.close();
}

void CUDA_Init(string &CentroidFileName, string &GraphFileName, int* &SectorTimeDict, double* &device_centroids_x, double* &device_centroids_y, int* &device_arrSizes, GraphNode** &device_graph, int* &device_Paths, int* &device_Paths_size, double* &device_Fitness, int* &device_Output, int* &device_Output_size,int NumSectors, int PopulationSize, int NumODPairs)
{
	//ONE TIME
	GraphNode* host_graph[NumSectors];
	int host_arrSizes[NumSectors];
	double host_centroids_x[NumSectors];
	double host_centroids_y[NumSectors];
	readGraph(GraphFileName,host_graph,host_arrSizes);
	readCentroids(CentroidFileName,host_centroids_x,host_centroids_y);
	hipMalloc((void **)&device_centroids_x, sizeof(double)*NumSectors);
	hipMalloc((void **)&device_centroids_y, sizeof(double)*NumSectors);
	hipMemcpy(device_centroids_x, host_centroids_x, sizeof(double)*(NumSectors),hipMemcpyHostToDevice);
	hipMemcpy(device_centroids_y, host_centroids_y, sizeof(double)*(NumSectors),hipMemcpyHostToDevice);
	hipMalloc((void **)&device_arrSizes, sizeof(int)*NumSectors);	
	hipMemcpy(device_arrSizes, host_arrSizes, sizeof(int)*(NumSectors),hipMemcpyHostToDevice);
	hipMalloc((void ***)&(device_graph), sizeof(GraphNode*)*NumSectors);
	for(int i=0;i<NumSectors;i++)
	{
		GraphNode* temp;
		hipMalloc((void**)&temp, sizeof(GraphNode) * host_arrSizes[i]);
		hipMemcpy(temp, host_graph[i], sizeof(GraphNode) * host_arrSizes[i], hipMemcpyHostToDevice);
		hipMemcpy(device_graph+i, &temp, sizeof(GraphNode*), hipMemcpyHostToDevice);
	}
	hipMalloc((void**)&SectorTimeDict,sizeof(int)*NumSectors*NumSectors);
	hipMemset(SectorTimeDict,0,sizeof(int)*NumSectors*NumSectors);
	hipMalloc((void**)&device_Output,sizeof(int)*NumODPairs*NumSectors);
	hipMemset(device_Output,-1,sizeof(int)*NumODPairs*NumSectors);
	hipMalloc((void **)&device_Output_size, sizeof(int)*NumODPairs);
	hipMemset(device_Output_size,0,sizeof(int)*NumODPairs);
	
	//RESET PER OD PAIR
	hipMallocManaged((void **)&(device_Paths), sizeof(int)*PopulationSize*MaxPathLen);
	hipMemset(device_Paths,-1,sizeof(int)*PopulationSize*MaxPathLen);
	hipMallocManaged((void **)&(device_Paths_size), sizeof(int)* PopulationSize);
	hipMemset(device_Paths_size,0,sizeof(int)* PopulationSize);
	hipMalloc((void **)&device_Fitness, sizeof(double)*PopulationSize);
	hipMemset(device_Fitness,-1,sizeof(double)*PopulationSize);
}

__global__ void update_SectorTimeDict(int* SectorTimeDict, int* device_Output, int* device_Output_size)
{
	int thread= threadIdx.x+(blockIdx.x*blockDim.x);
	if(thread < *device_Output_size)
	{
		int sector = device_Output[thread];
		SectorTimeDict[sector*MaxPathLen+thread]+=1;
	}
}

void getPaths(vector<pair<int,int>> &ODPairs, int Paths[][MaxPathLen], int NumSectors, int PopulationSize, int SelectionSize, int NumberOfMutations, int NumberOfGenerations, string& GraphFileName, string& CentroidFileName)
{	
	int* SectorTimeDict; //2D
	double* device_centroids_x;
	double* device_centroids_y;
	int *device_arrSizes;
	GraphNode** device_graph;
	int* device_Paths; //2D
	int* device_Paths_size;
	double* device_Fitness;
	int* device_Output; //2D
	int* device_Output_size;
	int NumODPairs=ODPairs.size();
	CUDA_Init(CentroidFileName, GraphFileName, SectorTimeDict, device_centroids_x, device_centroids_y, device_arrSizes, device_graph, device_Paths, device_Paths_size, device_Fitness, device_Output, device_Output_size ,NumSectors, PopulationSize,NumODPairs);
	for(int i=0;i<NumODPairs;i++)
	{
		int* output_path_ptr=device_Output+(i*NumSectors);
		int* output_path_size_ptr=device_Output_size+i;	GeneticAlgorithm(NumSectors,PopulationSize,SelectionSize,NumberOfMutations,NumberOfGenerations,ODPairs[i].first,ODPairs[i].second,SectorTimeDict,device_centroids_x,device_centroids_y,device_arrSizes,device_graph,device_Paths,device_Fitness,output_path_ptr,output_path_size_ptr,device_Paths_size);
		hipMemset(device_Paths,-1,sizeof(int)*PopulationSize*MaxPathLen);
		hipMemset(device_Paths_size,0,sizeof(int)* PopulationSize);
		hipMemset(device_Fitness,-1,sizeof(double)*PopulationSize);
		update_SectorTimeDict<<<1,NumThreads>>>(SectorTimeDict, output_path_ptr, output_path_size_ptr);
		hipDeviceSynchronize();
	}
	for(int i=0;i<NumODPairs;i++)
		hipMemcpy(Paths[i],device_Paths+i,MaxPathLen*sizeof(int),hipMemcpyDeviceToHost);
}

void readInput(vector<pair<int,int>>& ODPairs, string InputFileName)
{
	fstream file(InputFileName);
	string line="";
	vector<string> tokens;
	while(getline(file,line))
	{
		tokens.clear();
		tokenize(line,',',tokens);
		ODPairs.push_back({stoi(tokens[0]),stoi(tokens[1])});
	}
	file.close();
}

__device__ double getAngle(int A, int B, int C,double* device_centroids_x, double* device_centroids_y)
{
	double a = atan2(-(device_centroids_y[B]-device_centroids_y[A]),device_centroids_x[B]-device_centroids_x[A])*RadConvFactorToMultiply;
	double b = atan2(-(device_centroids_y[B]-device_centroids_y[C]),device_centroids_x[B]-device_centroids_x[C])*RadConvFactorToMultiply;
	if(abs(b-a)>180)
		return 180-(360-abs(b-a));
	else
		return 180-(abs(b-a));
}

__device__ void PathFitness(double* device_Fitness, int* device_Paths, int* device_Paths_size, int thread,GraphNode** device_graph,int* device_arrSizes, double* device_centroids_x, double* device_centroids_y)
{
	double angle = 1;
	double path_length=0;
	for (int i=1;i<device_Paths_size[thread];i++)
	{
		int cur=device_Paths[thread*MaxPathLen+(i-1)];
		int to = device_Paths[thread*MaxPathLen+(i)];
		bool found=false;
		for(int j=0;j<device_arrSizes[cur];j++)
		{
			if(to==device_graph[cur][j].vertexID)
			{
				path_length+=device_graph[cur][j].weight;
				found=true;
				break;
			}
		}
		if(!found)
		{
			printf("INVALID PATH\t%d\t%d\n",cur,to);
			return;
		}
	}
	device_Fitness[thread]=path_length;
	for (int i=0;i<device_Paths_size[thread]-2;i++)
		angle+=getAngle(device_Paths[thread*MaxPathLen+i],device_Paths[thread*MaxPathLen+(i+1)],device_Paths[thread*MaxPathLen+(i+2)],device_centroids_x,device_centroids_y);
}

__global__ void getInitPopulation(GraphNode** device_graph, int* device_arrSizes, int* device_Paths, int* device_Paths_size, double*device_Fitness, int start, int end, int PopulationSize,int seed,double* device_centroids_x, double* device_centroids_y)
{
	int thread= threadIdx.x+(blockIdx.x*blockDim.x);
	if(thread<PopulationSize)
	{	
		hiprandState_t state;
		hiprand_init(seed, thread, 0, &state);

		int ptr_pos=0;
		device_Paths[thread*MaxPathLen+ptr_pos++]=start;
		bool InitPath=false;
		bool visited[1250];
		int validIndex[20];
		int validIndexSize=0;
		int num_neighbors;
		int cur;
		while(!InitPath)
		{
			memset(visited,0,1250);
			visited[start]=true;
			ptr_pos=0;
			device_Paths[thread*MaxPathLen+ptr_pos++]=start;
			cur=start;
			while(!InitPath)
			{
				validIndexSize=0;
				num_neighbors=device_arrSizes[cur];
				for(int i=0;i<num_neighbors;i++)
				{
					if(!visited[device_graph[cur][i].vertexID])
						validIndex[validIndexSize++]=i;
				}
				if(validIndexSize==0)
					break;
				else
				{
					cur=device_graph[cur][validIndex[hiprand(&state)%validIndexSize]].vertexID;
					visited[cur]=true;
					device_Paths[thread*MaxPathLen+ptr_pos++]=cur;
					if(cur==end)
						InitPath=true;

				}

			}
		}
		device_Paths_size[thread]=ptr_pos;
		PathFitness(device_Fitness,device_Paths,device_Paths_size,thread,device_graph,device_arrSizes,device_centroids_x,device_centroids_y);
	}
}

void readCentroids(string CentroidFileName, double host_centroids_x[], double host_centroids_y[])
{
	string line="";
	fstream file(CentroidFileName);
	int sectorNum=0;
	vector<string> tokens;
	while(getline(file,line))
	{
		tokens.clear();
		tokenize(line,',',tokens);
		host_centroids_x[sectorNum]=stod(tokens[0]);
		host_centroids_y[sectorNum++]=stod(tokens[1]);
	}
	file.close();
}
void GeneticAlgorithm(int NumSectors,int PopulationSize, int SelectionSize, int NumberOfMutations, int NumberOfGenerations, int Start, int End, int* &SectorTimeDict, double* &device_centroids_x, double* &device_centroids_y, int* &device_arrSizes, GraphNode** &device_graph, int* &device_Paths, double* &device_Fitness, int* &device_Output, int* &device_Output_size, int* & device_Paths_size)
{	
	getInitPopulation<<<(PopulationSize/NumThreads)+1,NumThreads>>> (device_graph,device_arrSizes,device_Paths,device_Paths_size,device_Fitness,Start,End,PopulationSize,time(NULL),device_centroids_x,device_centroids_y);
	hipDeviceSynchronize();
}

void readGraph(string GraphFileName,GraphNode* host_graph[], int* arrSizes)
{
	string line="";
	fstream file(GraphFileName);
	vector<string> tokens;
	vector<string> pairString;
	int VNum=0;
	while(getline(file,line))
	{
		tokens.clear();
		tokenize(line,' ',tokens);
		int StartSec=stoi(tokens[0]);
		GraphNode* Neighbors = (GraphNode*)malloc(sizeof(GraphNode)*tokens.size()-1); 
		for(int i=1;i<tokens.size();i++)
		{
			pairString.clear();
			tokenize(tokens[i],',',pairString);
			GraphNode* node = new GraphNode;
			node->vertexID=stoi(pairString[0]);
			node->weight=stod(pairString[1]);
			Neighbors[i-1]=*node;
		}
		host_graph[VNum]=Neighbors;
		arrSizes[VNum]=tokens.size()-1;
		VNum++;
	}
	file.close();	
}

void tokenize(string &str, char delim, vector<string> &out)
{
	size_t start;
	size_t end = 0;
	while ((start = str.find_first_not_of(delim, end)) != string::npos)
	{
		end = str.find(delim, start);
		string s=str.substr(start, end - start);
		out.push_back(s);
	}
}
